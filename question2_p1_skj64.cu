
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <math.h>
#include <omp.h> 

//See values of N in assignment instructions.
#define N 10000

using namespace std;

//Do not change the seed, or your answer will not be correct
#define SEED 72

//For GPU implementation
#define BLOCKSIZE 1024


struct pointData{
double x;
double y;
};



void generateDataset(struct pointData * data);
__global__ void calcDistances(struct pointData * inputData, double * inputEpsilon, unsigned long long int  * numOfDistancesWithinEps);

int main(int argc, char *argv[])
{
	
	//Read epsilon distance from command line
	if (argc!=2)
	{
	printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
	return 0;
	}
	
	
	char inputEpsilon[20];
	strcpy(inputEpsilon,argv[1]);
	double epsilon=atof(inputEpsilon);
	
	

	//generate dataset:
	struct pointData * data;
	data=(struct pointData*)malloc(sizeof(struct pointData)*N);
	printf("\nSize of dataset (MiB): %f",(2.0*sizeof(double)*N*1.0)/(1024.0*1024.0));
	generateDataset(data);

	omp_set_num_threads(1);
	
	///////////////////Time set ups
	double totalTranferingTimeFromCPUToGPU;
	double startTransferTimeGPU;
	double endTransferTimeGPU;
	
	double totalTransferingTimeFromGPUToCPU;
	double startTransferCPU;
	double endTransferCPU;
	
	double totalTransferTime;
	
	double totalGPUKernelTime;
	double startGPUKern;
	double endGPUKern;
	///////////////////////////////////
	
	double tstart=omp_get_wtime();
	

	//Write your code here:
	//The data you need to use is stored in the variable "data", 
	//which is of type pointData
	
	hipError_t errCode=hipSuccess;
	
	if(errCode != hipSuccess)
	{
		cout << "\nLast error: " << errCode << endl; 	
	}
	
	struct pointData * dev_Data;
	double * dev_Epsilon;
	unsigned long long int * countOfPointInEpsilon;
	unsigned long long int * dev_countOfPointInEpsilon;
	countOfPointInEpsilon = (unsigned long long int *)malloc(sizeof(unsigned long long int));
	dev_countOfPointInEpsilon = (unsigned long long int *)malloc(sizeof(unsigned long long int));
	*countOfPointInEpsilon = 0;
	
	
	startTransferTimeGPU = omp_get_wtime();
	
	//allocate on the device: data
	errCode=hipMalloc((struct pointData**)&dev_Data, sizeof(struct pointData)*N);	
	if(errCode != hipSuccess) {
	cout << "\nError: point list error with code " << errCode << endl; 
	}
	
	//allocate epsilon value on device
	errCode=hipMalloc((double**)&dev_Epsilon, sizeof(double));
    if(errCode != hipSuccess) {
    cout << "\nError: B error with code " << errCode << endl;
    }
	
	//allocate the number of points in the epsilon on device
	errCode=hipMalloc((unsigned long long int**)&dev_countOfPointInEpsilon, sizeof(unsigned long long int));	
	if(errCode != hipSuccess) {
	cout << "\nError: points in Epsilon error with code " << errCode << endl; 
	}
	
	errCode=hipMemcpy( dev_Data, data, sizeof(struct pointData)*N, hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: dev_Data memcpy error with code " << errCode << endl; 
	}
	
	errCode=hipMemcpy( dev_Epsilon, &epsilon, sizeof(double), hipMemcpyHostToDevice);
    if(errCode != hipSuccess) {
    cout << "\nError: B memcpy error with code " << errCode << endl;
    }
	
	errCode=hipMemcpy( dev_countOfPointInEpsilon, countOfPointInEpsilon, sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess) {
	cout << "\nError: count in circle memcpy error with code " << errCode << endl; 
	}
	hipDeviceSynchronize();
	
	endTransferTimeGPU = omp_get_wtime();
	
	totalTranferingTimeFromCPUToGPU = endTransferTimeGPU - startTransferTimeGPU;
	
	printf("\nTotal time to transfer from CPU to GPU(s): %f",totalTranferingTimeFromCPUToGPU);
	
	
	startGPUKern = omp_get_wtime();
	
	const unsigned int totalBlocks=ceil(N*1.0/1024.0);
	printf("\ntotal blocks: %d",totalBlocks);
	calcDistances<<<totalBlocks,1024>>>(dev_Data, dev_Epsilon, dev_countOfPointInEpsilon);
	hipDeviceSynchronize();
	
	endGPUKern = omp_get_wtime();
	
	totalGPUKernelTime = endGPUKern - startGPUKern;
	
	printf("\nTotal time for GPU Kernel(s): %f",totalGPUKernelTime);
	
	
	startTransferCPU = omp_get_wtime();
	
	errCode=hipMemcpy( countOfPointInEpsilon, dev_countOfPointInEpsilon, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    if(errCode != hipSuccess) {
    cout << "\nError: getting result form GPU error with code " << errCode << endl;
    }
    hipDeviceSynchronize();
    
    endTransferCPU = omp_get_wtime();
    
    totalTransferingTimeFromGPUToCPU = endTransferCPU - startTransferCPU;
    
    printf("\nTotal number of points within epsilon (GPU): %llu",*countOfPointInEpsilon);
	
	hipDeviceSynchronize();

	double tend=omp_get_wtime();
	
	printf("\nTotal time (s): %f",tend-tstart);
	
	totalTransferTime = totalTranferingTimeFromCPUToGPU + totalTransferingTimeFromGPUToCPU;
	
	printf("\nTotal transfer time (s): %f",totalTransferTime);
	
	free(data);
	printf("\n");
	return 0;
}

__global__ void calcDistances(struct pointData * inputData, double * inputEpsilon, unsigned long long int  * numOfDistancesWithinEps) {

unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x);
if (tid>=N){
    return;
}
    
int dataIndex;
for (dataIndex = 0; dataIndex < N; dataIndex++){
    if ((sqrt(((inputData[tid].x - inputData[dataIndex].x) * (inputData[tid].x - inputData[dataIndex].x)) + 
        ((inputData[tid].y - inputData[dataIndex].y) * (inputData[tid].y - inputData[dataIndex].y)))) <= *inputEpsilon){
        atomicAdd(numOfDistancesWithinEps, int(1));
    }
}

return;
}

//Do not modify the dataset generator or you will get the wrong answer
void generateDataset(struct pointData * data)
{

	//seed RNG
	srand(SEED);


	for (unsigned int i=0; i<N; i++){
		data[i].x=1000.0*((double)(rand()) / RAND_MAX);	
		data[i].y=1000.0*((double)(rand()) / RAND_MAX);	
	}
	

}

void warmUpGPU(){
hipDeviceSynchronize();
return;
}